#include "hip/hip_runtime.h"
#include "nbody_cuda.cuh"
#include <cmath>
#include <iostream>

#define BLOCK_SIZE 128

namespace
{
constexpr float G = 6.67408e-11;

struct DVector
{
	int N;
	float *data;
};

__device__
void body_body_interaction(float &ax, float &ay, float &az,
		                   float x0, float y0, float z0,
						   float x1, float y1, float z1, float m1, float softening_squared )
{
	float dx = x0 - x1;
	float dy = y0 - y1;
	float dz = z0 - z1;

	float d_sqr = dx*dx + dy*dy + dz*dz + softening_squared;
	float inv = 1./sqrt(d_sqr);
	float inv3 = inv*inv*inv;

	float s = m1 * inv3;
	ax = dx * s;
	ay = dy * s; 
    az = dz * s;
}
}

__global__
void compute_graviation_cuda(float *force, const float *pos_mass, float softening_squared, int N)
{

	float4 *pos_mass4 = (float4 *)pos_mass;
	__shared__ float4 buffer[BLOCK_SIZE];

	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i+=blockDim.x*gridDim.x)
	{
		float acc[3] = {0.f, 0.f, 0.f};
		float4 me = pos_mass4[i];
#pragma unroll 32
		for (int j = 0; j<N/4; j+=  blockDim.x)
		{
			buffer[threadIdx.x] = pos_mass4[0+threadIdx.x];
			__syncthreads();

			for (int k = 0; k < N; k++){

				float4 body = buffer[k];
				float fx=0, fy=0, fz=0;

				// force j on i (f_{ij})
				body_body_interaction(fx, fy, fz,
									  me.x, me.y, me.z,
									  body.x, body.y, body.z, body.w, softening_squared );

				acc[0]+=fx;
				acc[1]+=fy;
				acc[2]+=fz;
			}
            __syncthreads();
		}
        force[3*i + 0] = acc[0];
        force[3*i + 1] = acc[1];
        force[3*i + 2] = acc[2];
	}
}

DVector make_dvector(const std::vector<float>& V)
{
	DVector dV;
    dV.N = int(V.size());

	float *dV_ptr;
	hipMalloc(&dV_ptr, dV.N*sizeof(float));
	hipMemcpy(dV_ptr, V.data(), dV.N*sizeof(float), hipMemcpyHostToDevice);
	dV.data = dV_ptr;
	return dV;
}

DVector make_dvector(int n)
{
	DVector dV;
	dV.N = n;

	float *dV_ptr;
	hipMalloc(&dV_ptr, n*sizeof(float));
	dV.data = dV_ptr;
	return dV;
}

std::vector<float> extract_vector(DVector dV)
{
	std::vector<float>v(dV.N);
	hipMemcpy(v.data(), dV.data, dV.N*sizeof(float), hipMemcpyDeviceToHost);
	return v;
}

std::vector<float> compute_graviation_gpu(const std::vector<float> &pos_mass, float softening_squared, int N)
{
	auto d_force = make_dvector(3*N);
	auto d_posmas = make_dvector(pos_mass);

	dim3 block(BLOCK_SIZE);
	dim3 grid((N + BLOCK_SIZE - 1)/ BLOCK_SIZE);
    compute_graviation_cuda<<<grid, block>>>(d_force.data, d_posmas.data, 0.3, N);
    auto force = extract_vector(d_force);
    return force;
}


